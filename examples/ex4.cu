#include "hip/hip_runtime.h"
#include <cstdlib>

#include "../src/tensor.cuh"

using namespace tnb;


template <typename TensorType>
__global__ void add_scalar(TensorType A, typename TensorType::value_type scalar) {
   index_t i = blockDim.x * blockIdx.x + threadIdx.x;
   for(; i < A.size(); i += gridDim.x * blockDim.x) {
      A[i] += scalar;
   }
}


template <typename T, index_t M>
bool verify(const T& x, const typename T::value_type (&scalars)[M]) {
   for(index_t i = 0; i < x.extent(0); ++i) {
      auto s = lslice(x, i);
      for(index_t k = 0; k < s.size(); ++k) {
         if(s[k] > scalars[i] + 1 || s[k] < scalars[i]) {
            return false;
         }
      }
   }
   return true;
}


int main() {
   using T = float;
   constexpr int M = 4;
   constexpr int N = 10000;
   const Extents<3> ext(M, N, N);
   const Extents<2> sub_ext(N, N);
   const auto nbytes = sub_ext.size() * sizeof(T);

   Tensor<T, 3, Pinned> x(ext);
   random(x);

   hipStream_t streams[M];
   T* xd[M]{};
   const T scalars[M]{1, 11, 101, 1001};

   timer t;
   for(int i = 0; i < M; ++i) {
      ASSERT_CUDA(hipStreamCreate(&streams[i]));
      ASSERT_CUDA(hipMallocAsync(&xd[i], nbytes, streams[i]));
      auto x_gpu = attach_device(xd[i], sub_ext);  // using xd[i] is safe

      x_gpu.copy_async(lslice(x, i), streams[i]);
      add_scalar<<<8, 8, 0, streams[i]>>>(x_gpu, scalars[i]);
      lslice(x, i).copy_async(x_gpu, streams[i]);

      ASSERT_CUDA(hipFreeAsync(xd[i], streams[i]));
   }

   for(int i = 0; i < M; ++i) {
      ASSERT_CUDA(hipStreamSynchronize(streams[i]));
      ASSERT_CUDA(hipStreamDestroy(streams[i]));
   }

   assert(verify(x, scalars));

   if(N <= 8) {
      std::cout << x << std::endl;
   }
   ASSERT_CUDA(hipDeviceReset());

   return EXIT_SUCCESS;
}
