#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>

#include "../src/tensor.cuh"

using namespace tnb;


template <typename T>
__global__ void fill(CudaMatrix<T> A) {
   auto rid = blockDim.x * blockIdx.x + threadIdx.x;
   for(; rid < A.extent(0); rid += gridDim.x * blockDim.x) {
      auto row = lslice(A, rid);
      for(auto& x : row) {
         x = rid;
      }
   }
}


int main() {
   CudaMatrix<float> A;
   A.Allocate(16, 8);
   fill<<<4, 4>>>(A);
   std::cout << A << std::endl;
   A.Free();

   return EXIT_SUCCESS;
}
