#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>

#include "../src/tensor.cuh"

using namespace tnb;


template <typename T>
__global__ void row_iota(CudaMatrix<T>* A) {
   auto rid = blockDim.x * blockIdx.x + threadIdx.x;
   for(; rid < A->extent(0); rid += gridDim.x * blockDim.x) {
      auto row = lslice(*A, rid);
      for(auto& x : row) {
         x = rid;
      }
   }
}


int main() {
   CudaMatrix<float> A(16, 8);
   row_iota<<<4, 4>>>(A.cuda_ptr());
   // hipDeviceSynchronize is not used since output operator implicitly synchronizes.
   std::cout << A << std::endl;
   return EXIT_SUCCESS;
}
