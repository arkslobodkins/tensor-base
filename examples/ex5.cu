#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>

#include "../src/tensor.cuh"

using namespace tnb;


template <typename T>
__global__ void row_iota(UnifiedMatrix<T>* A) {
   auto rid = blockDim.x * blockIdx.x + threadIdx.x;
   for(; rid < A->extent(0); rid += gridDim.x * blockDim.x) {
      auto row = lslice(*A, rid);
      for(auto& x : row) {
         x = rid;
      }
   }
}


int main() {
   UnifiedMatrix<float> A(16, 8);
   row_iota<<<4, 4>>>(A.cuda_ptr());
   hipDeviceSynchronize();
   for(index_t i = 0; i < A.extent(0); ++i) {
      A(i, 0) = 777;
   }
   std::cout << A << std::endl;
   return EXIT_SUCCESS;
}
